#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cmath>    // for fabsf
#include <fstream>  // for CSV output
#include <iostream>
#include <vector>

#define TOL 1e-5f
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    std::cerr << msg << " CUDA ERROR: " << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
}

void checkCublasError(hipblasStatus_t status, const char *msg) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << msg << " CUBLAS ERROR: " << status << std::endl;
    exit(EXIT_FAILURE);
  }
}

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void mysgemm_v4(int M, int N, int K, float alpha, float*A, float* B,
                            float beta, float *C) {

  int bx = blockIdx.x;
  int by = blockIdx.y;

  const int block_row_thread = BN / TN;
  const int block_col_thread = BM / TM;
  const int thread_num = block_row_thread * block_col_thread;
  
  //计算tile块在block中的二维坐标
  int ty = (threadIdx.x / block_row_thread) * TM;
  int tx = (threadIdx.x % block_row_thread) * TN;

  __shared__ float As[BK*BM];
  __shared__ float Bs[BK*BN];

  const int ldg_a_num = BK * BM / thread_num / 4;
  const int ldg_b_num = BK * BN / thread_num / 4;
  
  //计算tile中a_tile的行列起始位置
  const int a_tile_row = threadIdx.x / (BK / 4);
  const int a_tile_col = threadIdx.x % (BK / 4) * 4;
  int a_tile_stride = BM / ldg_a_num;
  const int b_tile_row = threadIdx.x / (BN / 4);
  const int b_tile_col = threadIdx.x % (BN / 4) * 4;
  int b_tile_stride = BK / ldg_b_num;

  float accum[TM][TN] = {0.};

  // float ldg_a_reg[4 * ldg_a_num] = {0.};
  float ldg_a_reg[4] = {0.};

  //读取共享内存时也是用向量化加载 利用SIMD
  float a_frag[TM];
  float b_frag[TN];
  
  //坐标变换分清楚row+col即可
  A = &A[by * K * BM];
  B = &B[bx * BN];
  C = &C[by * BM * N + bx * BN];
  

#pragma unroll
  for(int k = 0; k < K; k += BK) {
#pragma unroll
   //搬运数据到共享内存
   //对As转置后存储 以BK为行BM为列 减少后续计算时的cache miss
    // for(int i = 0; i < BM; i += a_tile_stride) {
    //   int ldg_index = i / a_tile_stride * 4;
    //   FETCH_FLOAT4(ldg_a_reg[ldg_index]) = 
    //     FETCH_FLOAT4(A[OFFSET(a_tile_row + i, a_tile_col, K)]);
    //   As[OFFSET(a_tile_col, a_tile_row + i, BM)] = ldg_a_reg[ldg_index];
    //   As[OFFSET(a_tile_col + 1, i + a_tile_row, BM)] = ldg_a_reg[1];
    //   As[OFFSET(a_tile_col + 2, i + a_tile_row, BM)] = ldg_a_reg[2];
    //   As[OFFSET(a_tile_col + 3, i + a_tile_row, BM)] = ldg_a_reg[3];
    // }
    for(int i = 0; i < BM; i += a_tile_stride) {
      // int ldg_index = i / a_tile_stride * 4;
      FETCH_FLOAT4(ldg_a_reg[0]) = 
        FETCH_FLOAT4(A[OFFSET(a_tile_row + i, a_tile_col, K)]);
      As[OFFSET(a_tile_col, a_tile_row + i, BM)] = ldg_a_reg[0];
      As[OFFSET(a_tile_col + 1, i + a_tile_row, BM)] = ldg_a_reg[1];
      As[OFFSET(a_tile_col + 2, i + a_tile_row, BM)] = ldg_a_reg[2];
      As[OFFSET(a_tile_col + 3, i + a_tile_row, BM)] = ldg_a_reg[3];
    }
#pragma unroll
    for(int i = 0; i < BK; i += b_tile_stride) {
      FETCH_FLOAT4(Bs[OFFSET(b_tile_row + i, b_tile_col, BN)]) = 
        FETCH_FLOAT4(B[OFFSET(b_tile_row + i, b_tile_col, N)]);
    }
    __syncthreads();
    A += BK;
    B += BK * N;
    //做运算
#pragma unroll
    for(int i = 0; i < BK; i++) {
#pragma unroll  //加载TM的一行数据
      for(int r = 0; r < TM; r += 4) {
        FETCH_FLOAT4(a_frag[r]) = FETCH_FLOAT4(As[OFFSET(i, ty + r, BM)]);
      }
#pragma unroll  //加载TN的一行数据
      for(int c = 0; c < TN; c += 4) {
        FETCH_FLOAT4(b_frag[c]) = FETCH_FLOAT4(Bs[OFFSET(i, tx + c, BN)]);
      }
#pragma unroll
      for(int m = 0; m < TM; m++) {
        for(int n = 0; n < TN; n++) {
          accum[m][n] += a_frag[m] * b_frag[n];
        }
      }
    }
    __syncthreads();
  }
#pragma unroll
  for(int m = 0; m < TM; m++) {
    for(int n = 0; n < TN; n += 4) {
      float4 ctmp = FETCH_FLOAT4(C[OFFSET(ty + m, tx + n, N)]);
      ctmp.x = alpha * accum[m][n] + beta * ctmp.x;
      ctmp.y = alpha * accum[m][n+1] + beta * ctmp.y;
      ctmp.z = alpha * accum[m][n+2] + beta * ctmp.z;
      ctmp.w = alpha * accum[m][n + 3] + beta * ctmp.w;
      FETCH_FLOAT4(C[OFFSET(ty + m, tx + n, N)]) = ctmp;
    }
  }  

}


#define CEIL_DIV(M, N) ((M) + (N) - 1) / (N)
int main() {
  int device_id = 1;
  checkCudaError(hipSetDevice(device_id), "hipSetDevice failed");
  std::vector<int> sizes = {128, 256, 512, 1024, 2048, 4096, 8192};
  // 打开CSV文件
  std::ofstream csv_file("sgemm_benchmark_v4_k8.csv");
  csv_file << "Size,CUBLAS_GFLOPS,MySGEMM_FLOPS,Matched" << std::endl;

  for (int N : sizes) {
    std::cout << "Testing size: " << N << std::endl;

    size_t size = N * N * sizeof(float);
    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C_cublas = (float *)malloc(size);
    float *C_v1 = (float *)malloc(size);

    float *d_A, *d_B, *d_C_v1;
    checkCudaError(hipMalloc(&d_A, size), "hipMalloc d_A failed");
    checkCudaError(hipMalloc(&d_B, size), "hipMalloc d_B failed");
    checkCudaError(hipMalloc(&d_C_v1, size), "hipMalloc d_C_v1 failed");

    bool out_of_memory = false;

    try {
      // 初始化矩阵 A 和 B
      for (int i = 0; i < N * N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
      }

      // 拷贝到设备
      checkCudaError(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice),
                     "hipMemcpy A to device failed");
      checkCudaError(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice),
                     "hipMemcpy B to device failed");

      hipblasHandle_t handle;
      checkCublasError(hipblasCreate(&handle), "hipblasCreate failed");
      float alpha = 1.0f;
      float beta = 0.0f;

      hipEvent_t start, stop;
      checkCudaError(hipEventCreate(&start), "cudaEventCreate(start) failed");
      checkCudaError(hipEventCreate(&stop), "cudaEventCreate(stop) failed");

      // warmup
      int warpup_time = 10;  // 热身次数
      for (int i = 0; i < warpup_time; ++i) {
        checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                                     &alpha, d_B, N, d_A, N, &beta, d_C_v1, N),
                         "hipblasSgemm failed");
      }
      hipDeviceSynchronize();

      // cuBLAS SGEMM
      int repeat_time = 5;
      checkCudaError(hipEventRecord(start),
                     "cudaEventRecord(start cublas) failed");
      for (int i = 0; i < repeat_time; ++i) {
        checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                                     &alpha, d_B, N, d_A, N, &beta, d_C_v1, N),
                         "hipblasSgemm failed");
      }

      checkCudaError(hipEventRecord(stop),
                     "cudaEventRecord(stop cublas) failed");
      checkCudaError(hipEventSynchronize(stop),
                     "hipEventSynchronize cublas failed");

      float cublas_time = 0;
      checkCudaError(hipEventElapsedTime(&cublas_time, start, stop),
                     "hipEventElapsedTime cublas failed");

      // 拷贝 cuBLAS 结果
      checkCudaError(hipMemcpy(C_cublas, d_C_v1, size, hipMemcpyDeviceToHost),
                     "hipMemcpy C_cublas failed");

      // mysgemm_v1
      checkCudaError(hipMemset(d_C_v1, 0, size), "hipMemset d_C_v1 failed");

      dim3 blockDim(256);
      dim3 gridDim(CEIL_DIV(N, 128), CEIL_DIV(N, 128));

      for (int i = 0; i < warpup_time; ++i) {
        mysgemm_v4<128, 128, 8, 8, 8>
            <<<gridDim, blockDim>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
      }

      hipDeviceSynchronize();
      checkCudaError(hipMemset(d_C_v1, 0, size), "hipMemset d_C_v1 failed");

      checkCudaError(hipEventRecord(start),
                     "cudaEventRecord(start v1) failed");

      for (int i = 0; i < repeat_time; ++i) {
        mysgemm_v4<128, 128, 8, 8, 8>
            <<<gridDim, blockDim>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
      }
      checkCudaError(hipEventRecord(stop), "cudaEventRecord(stop v1) failed");
      checkCudaError(hipEventSynchronize(stop),
                     "hipEventSynchronize v1 failed");
      float v1_time = 0;
      checkCudaError(hipEventElapsedTime(&v1_time, start, stop),
                     "hipEventElapsedTime v1 failed");

      // 拷贝手写 kernel 结果
      checkCudaError(hipMemcpy(C_v1, d_C_v1, size, hipMemcpyDeviceToHost),
                     "hipMemcpy C_v1 failed");
      // 结果比较
      int error_count = 0;
      for (int i = 0; i < N * N && error_count < 10; ++i) {
        if (fabsf(C_cublas[i] - C_v1[i]) > TOL) {
          error_count++;
        }
      }

      float cublas_gflops =
          repeat_time * 2.0f * N * N * N / (cublas_time * 1e6f);  // GFlops
      float v1_gflops =
          repeat_time * 2.0f * N * N * N / (v1_time * 1e6f);  // GFlops
      // 写入CSV
      csv_file << N << "," << cublas_gflops << "," << v1_gflops << ","
               << (error_count == 0 ? "1" : "0") << std::endl;

      // 释放资源
      hipblasDestroy(handle);
      hipEventDestroy(start);
      hipEventDestroy(stop);
      hipFree(d_A);
      hipFree(d_B);
      hipFree(d_C_v1);

      free(A);
      free(B);
      free(C_cublas);
      free(C_v1);

    } catch (...) {
      std::cerr << "Out of memory or error during testing size: " << N
                << std::endl;
      out_of_memory = true;
    }

    if (!out_of_memory) {
      std::cout << "Finished size: " << N << std::endl;
    } else {
      csv_file << N << ",OOM,OOM,0" << std::endl;
    }
  }

  csv_file.close();

  std::cout << "Benchmark completed. Results saved to 'sgemm_benchmark.csv'"
            << std::endl;
  return 0;
}
